#include "hip/hip_runtime.h"
/* Copyright (c) 2017-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */
#include <algorithm>
#include <array>
#include <climits>
#include <cmath>
#include <cstdio>
#include <iostream>
#include <iterator>
#include <sstream>

#include <omp.h>

#ifdef HAVE_CUB
#include <cub/block/block_reduce.cuh>
#endif  // HAVE_CUB

#ifdef USE_NVTX
#include <nvToolsExt.h>

const uint32_t colors[] = {0x0000ff00, 0x000000ff, 0x00ffff00, 0x00ff00ff,
                           0x0000ffff, 0x00ff0000, 0x00ffffff};
const int num_colors = sizeof(colors) / sizeof(uint32_t);

#define PUSH_RANGE(name, cid)                              \
    {                                                      \
        int color_id = cid;                                \
        color_id = color_id % num_colors;                  \
        nvtxEventAttributes_t eventAttrib = {0};           \
        eventAttrib.version = NVTX_VERSION;                \
        eventAttrib.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE;  \
        eventAttrib.colorType = NVTX_COLOR_ARGB;           \
        eventAttrib.color = colors[color_id];              \
        eventAttrib.messageType = NVTX_MESSAGE_TYPE_ASCII; \
        eventAttrib.message.ascii = name;                  \
        nvtxRangePushEx(&eventAttrib);                     \
    }
#define POP_RANGE nvtxRangePop();
#else
#define PUSH_RANGE(name, cid)
#define POP_RANGE
#endif

#define CUDA_RT_CALL(call)                                                                  \
    {                                                                                       \
        hipError_t cudaStatus = call;                                                      \
        if (hipSuccess != cudaStatus)                                                      \
            fprintf(stderr,                                                                 \
                    "ERROR: CUDA RT call \"%s\" in line %d of file %s failed "              \
                    "with "                                                                 \
                    "%s (%d).\n",                                                           \
                    #call, __LINE__, __FILE__, hipGetErrorString(cudaStatus), cudaStatus); \
    }

typedef float real;
constexpr real tol = 1.0e-8;

const real PI = 2.0 * std::asin(1.0);

__global__ void initialize_boundaries(real* __restrict__ const a_new, real* __restrict__ const a,
                                      const real pi, const int offset, const int nx,
                                      const int my_ny, const int ny) {
    for (int iy = blockIdx.x * blockDim.x + threadIdx.x; iy < my_ny; iy += blockDim.x * gridDim.x) {
        const real y0 = sin(2.0 * pi * (offset + iy) / (ny - 1));
        a[iy * nx + 0] = y0;
        a[iy * nx + (nx - 1)] = y0;
        a_new[iy * nx + 0] = y0;
        a_new[iy * nx + (nx - 1)] = y0;
    }
}

template <int BLOCK_DIM_X, int BLOCK_DIM_Y>
__global__ void jacobi_kernel(real* __restrict__ const a_new, const real* __restrict__ const a,
                              real* __restrict__ const l2_norm, const int iy_start,
                              const int iy_end, const int nx, const bool calculate_norm) {
#ifdef HAVE_CUB
    typedef hipcub::BlockReduce<real, BLOCK_DIM_X, hipcub::BLOCK_REDUCE_WARP_REDUCTIONS, BLOCK_DIM_Y>
        BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;
#endif  // HAVE_CUB
    int iy = blockIdx.y * blockDim.y + threadIdx.y + iy_start;
    int ix = blockIdx.x * blockDim.x + threadIdx.x + 1;
    real local_l2_norm = 0.0;

    if (iy < iy_end && ix < (nx - 1)) {
        const real new_val = 0.25 * (a[iy * nx + ix + 1] + a[iy * nx + ix - 1] +
                                     a[(iy + 1) * nx + ix] + a[(iy - 1) * nx + ix]);
        a_new[iy * nx + ix] = new_val;
        if (calculate_norm) {
            real residue = new_val - a[iy * nx + ix];
            local_l2_norm += residue * residue;
        }
    }
    if (calculate_norm) {
#ifdef HAVE_CUB
        real block_l2_norm = BlockReduce(temp_storage).Sum(local_l2_norm);
        if (0 == threadIdx.y && 0 == threadIdx.x) atomicAdd(l2_norm, block_l2_norm);
#else
        atomicAdd(l2_norm, local_l2_norm);
#endif  // HAVE_CUB
    }
}

template <typename T>
T get_argval(char** begin, char** end, const std::string& arg, const T default_val) {
    T argval = default_val;
    char** itr = std::find(begin, end, arg);
    if (itr != end && ++itr != end) {
        std::istringstream inbuf(*itr);
        inbuf >> argval;
    }
    return argval;
}

bool get_arg(char** begin, char** end, const std::string& arg) {
    char** itr = std::find(begin, end, arg);
    if (itr != end) {
        return true;
    }
    return false;
}

double single_gpu(const int nx, const int ny, const int iter_max, real* const a_ref_h,
                  const int nccheck, const bool print);

int main(int argc, char* argv[]) {
    const int iter_max = get_argval<int>(argv, argv + argc, "-niter", 1000);
    const int nccheck = get_argval<int>(argv, argv + argc, "-nccheck", 1);
    const int nx = get_argval<int>(argv, argv + argc, "-nx", 16384);
    const int ny = get_argval<int>(argv, argv + argc, "-ny", 16384);
    const bool csv = get_arg(argv, argv + argc, "-csv");

    if (nccheck != 1) {
        fprintf(stderr, "Only nccheck = 1 is supported\n");
        return -1;
    }

    CUDA_RT_CALL(hipSetDevice(0));
    CUDA_RT_CALL(hipFree(0));

    real* a_ref_h;
    CUDA_RT_CALL(hipHostMalloc(&a_ref_h, nx * ny * sizeof(real)));
    
    double runtime_serial = single_gpu(nx, ny, iter_max, a_ref_h, nccheck, !csv);

    if (csv) {
        printf("single_gpu, %d, %d, %d, %d, %f\n", nx, ny, iter_max, nccheck, runtime_serial);
    } else {
        printf("%dx%d: 1 GPU: %8.4f s\n", ny, nx, runtime_serial);
    }

    return 0;
}

double single_gpu(const int nx, const int ny, const int iter_max, real* const a_ref_h,
                  const int nccheck, const bool print) {
    real* a;
    real* a_new;

    real* l2_norm_d;
    real* l2_norm_h;

    int iy_start = 1;
    int iy_end = (ny - 1);

    CUDA_RT_CALL(hipMalloc(&a, nx * ny * sizeof(real)));
    CUDA_RT_CALL(hipMalloc(&a_new, nx * ny * sizeof(real)));

    CUDA_RT_CALL(hipMemset(a, 0, nx * ny * sizeof(real)));
    CUDA_RT_CALL(hipMemset(a_new, 0, nx * ny * sizeof(real)));

    // Set diriclet boundary conditions on left and right boarder
    initialize_boundaries<<<ny / 128 + 1, 128>>>(a, a_new, PI, 0, nx, ny, ny);
    CUDA_RT_CALL(hipGetLastError());
    CUDA_RT_CALL(hipDeviceSynchronize());

    CUDA_RT_CALL(hipMalloc(&l2_norm_d, sizeof(real)));
    CUDA_RT_CALL(hipHostMalloc(&l2_norm_h, sizeof(real)));

    CUDA_RT_CALL(hipDeviceSynchronize());

    if (print)
        printf(
            "Single GPU jacobi relaxation: %d iterations on %d x %d mesh with "
            "norm "
            "check every %d iterations\n",
            iter_max, ny, nx, nccheck);

    constexpr int dim_block_x = 32;
    constexpr int dim_block_y = 32;
    dim3 dim_grid((nx + dim_block_x - 1) / dim_block_x, (ny + dim_block_y - 1) / dim_block_y, 1);

    int iter = 0;
    bool calculate_norm;
    real l2_norm = 1.0;

    double start = omp_get_wtime();
    PUSH_RANGE("Jacobi solve", 0)
    while (l2_norm > tol && iter < iter_max) {
        CUDA_RT_CALL(hipMemset(l2_norm_d, 0, sizeof(real)));

        calculate_norm = (iter % nccheck) == 0 || (print && ((iter % 100) == 0));
        jacobi_kernel<dim_block_x, dim_block_y>
            <<<dim_grid, {dim_block_x, dim_block_y, 1}, 0, 0>>>(
                a_new, a, l2_norm_d, iy_start, iy_end, nx, calculate_norm);
        CUDA_RT_CALL(hipGetLastError());

        if (calculate_norm) {
            CUDA_RT_CALL(hipMemcpy(l2_norm_h, l2_norm_d, sizeof(real), hipMemcpyDeviceToHost));
        }

        // Apply periodic boundary conditions

        CUDA_RT_CALL(hipMemcpy(a_new, a_new + (iy_end - 1) * nx, nx * sizeof(real),
                                     hipMemcpyDeviceToDevice));
        CUDA_RT_CALL(hipMemcpy(a_new + iy_end * nx, a_new + iy_start * nx, nx * sizeof(real),
                                     hipMemcpyDeviceToDevice));

        if (calculate_norm) {
	    CUDA_RT_CALL(hipDeviceSynchronize());
            //CUDA_RT_CALL(hipStreamSynchronize(compute_stream));
            l2_norm = *l2_norm_h;
            l2_norm = std::sqrt(l2_norm);
            if (print && (iter % 100) == 0) printf("%5d, %0.6f\n", iter, l2_norm);
        }

        std::swap(a_new, a);
        iter++;
    }
    POP_RANGE
    double stop = omp_get_wtime();

    CUDA_RT_CALL(hipMemcpy(a_ref_h, a, nx * ny * sizeof(real), hipMemcpyDeviceToHost));

    CUDA_RT_CALL(hipHostFree(l2_norm_h));
    CUDA_RT_CALL(hipFree(l2_norm_d));

    CUDA_RT_CALL(hipFree(a_new));
    CUDA_RT_CALL(hipFree(a));
    return (stop - start);
}

