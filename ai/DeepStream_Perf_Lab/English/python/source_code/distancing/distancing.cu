#include "hip/hip_runtime.h"
#include <pybind11/stl.h>
#include <thread>
#include <vector>
#include <cmath>

using namespace std;
namespace py = pybind11;
typedef tuple<float, float, float> centroid;

float compute_dist(centroid& p1, centroid& p2)
{
    float x1, y1, h1, x2, y2, h2;
    std::tie(x1, y1, h1) = p1;
    std::tie(x2, y2, h2) = p2;
    float dx = x2 - x1;
    float dy = y2 - y1;

    float lx = dx * 170 * (1/h1 + 1/h2) / 2;
    float ly = dy * 170 * (1/h1 + 1/h2) / 2;

    float l = sqrt(lx*lx + ly*ly);
    return l;
}

float compute_min_dist(int p, centroid& point, vector<centroid>& points) 
{
    vector<float> distances;
    for (auto & p2 : points) {
        distances.push_back(compute_dist(point, p2));
    }
    distances[p] = 1000000.0;
    float min_dist = *std::min_element(distances.begin(), distances.end());
    return min_dist;
}

vector<float> get_min_distances(vector<centroid>& points)
{
    vector<float> out;
    for (int p = 0; p < points.size(); p++) {
        float min_dist = compute_min_dist(p, points[p], points);
        out.push_back(min_dist);
    }
    return out;
}

PYBIND11_MODULE(distancing, m) {
    m.def("get_min_distances", &get_min_distances, "Get min distances");
}
